
#include <hip/hip_runtime.h>
#include <stdio.h>


void check(hipError_t err) {
	if (err != hipSuccess)
        	printf("The error is %s.\n", hipGetErrorString(err));
}

__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(void) {
	int a, b, c;              // host copies of a, b, c
	int *d_a, *d_b, *d_c;     // device copies of a, b, c
	int size = sizeof(int);

	// Allocate space for device copies of a, b, c
	check((hipError_t)hipMalloc((void **)&d_a, size));
	check((hipError_t)hipMalloc((void **)&d_b, size));
	check((hipError_t)hipMalloc((void **)&d_c, size));

	// Setup input valies
	a = 12;
	b = 7;

	// Copy inputs to device
	check((hipError_t)hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice));
	check((hipError_t)hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice));

	// Launch all() kernel on GPU
	add<<<1,1>>>(d_a, d_b, d_c);

	// Copy result back to host
	check((hipError_t)hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost));

	printf("%d\n", c);

	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	return 0;
}

