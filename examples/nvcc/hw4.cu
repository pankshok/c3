
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 16

void check(hipError_t err) {
	if (err != hipSuccess)
		printf("The error is %s.\n", hipGetErrorString(err));
}
		
void print_array(int *arr, int len) {
	for (int row = 0; row < 16; row++) {
		for (int i = 0; i < len; i++)
			printf("%d  ", arr[i]);
		printf("\n");
	}
}

void random_ints(int *a, int n) {
	int i;
	for (i = 0; i < n; i++)
		a[i] = (int)(rand() / (RAND_MAX / 1.5));
}


int main(void) {
	int *a, *b, *c;              // host copies of a, b, c
	int *d_a, *d_b, *d_c;        // device copies of a, b, c
	int size = N * sizeof(int);


	// Allocate space for device copies of a, b, c
	check((hipError_t)hipMalloc((void **)&d_a, size));
	check((hipError_t)hipMalloc((void **)&d_b, size));
	check((hipError_t)hipMalloc((void **)&d_c, size));

	// Allocate space for host copies of a, b, c and setup input values
	a = (int*)malloc(size); random_ints(a, N);
	b = (int*)malloc(size); random_ints(b, N);
	c = (int*)malloc(size);

	// Copy inputs to device
	check((hipError_t)hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
	check((hipError_t)hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));

	// Launch all() kernel on GPU
	add<<<N,1>>>(d_a, d_b, d_c);

	// Copy result back to host
	check((hipError_t)hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));

	// Print results
	printf("Array a:\n");
	print_array(a, N);
	printf("Array b:\n");
	print_array(b, N);
	printf("Sum of a and b:\n");
	print_array(c, N);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	return 0;
}

