
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512

void random_ints(int *a, int n) {
	int i;
	for (i = 0; i < n; i++)
		a[i] = rand();
}


int main(void) {
	int *a, *b, *c;              // host copies of a, b, c
	int *d_a, *d_b, *d_c;        // device copies of a, b, c
	int size = N * sizeof(int);


	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Allocate space for host copies of a, b, c and setup input values
	a = (int*)malloc(size); random_ints(a, N);
	b = (int*)malloc(size); random_ints(a, N);
	c = (int*)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch all() kernel on GPU
	add<<<N,1>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	return 0;
}

